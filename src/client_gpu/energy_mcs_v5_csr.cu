#include "hip/hip_runtime.h"
// sparse matrix in CSR format



float elhm = 0.0f;
int i = threadIdx.x;



int elhm2 = ncol[...];


__shared__ float elhm1s[1024];

//for (int i = threadIdx.x; i < npoint; i += blockDim.x) { // blockDim.x

if (i < npoint) {
  const int l = mcsm->idx_col[i];
  const float dx = lig_x2[l] - mcsm->x[i]; // do not use __LDG
  const float dy = lig_y2[l] - mcsm->y[i];
  const float dz = lig_z2[l] - mcsm->z[i];
  elhm1s[i] = dx * dx + dy * dy + dz * dz;
}


__syncthreads ();
if (i < nrow) {
  float elhm1 = 0.0f;
  for (int j = 0; j < mcsm->ncol[i]; ++j) {
    elhm1 += elhm1s[mcsm->row_ptr + j];
  }
  elhm += mcs_tcc[m] * sqrtf (elhm1 / (float) elhm2);
 }



BlockReduceSum_1_d_2 (elhm);
if (threadIdx.x == 0)
  e_s[7] = logf (elhm / mcs_nrow);


