#include "hip/hip_runtime.h"

// parse matrix
// no performance difference
// result is wrong



float elhm = 0.0f;
ty = threadIdx.x / bdx_mcs;
tx = threadIdx.x % bdx_mcs;


//Mcs *mcs_ = &complex->mcs[0];

// lhm loop, ~11
// unrolling outer loop does not help
//#pragma unroll 4
for (int j = 0; j < mcs_nrow; j += bdy_mcs) { // y loop
  float elhm1 = 0.0f;
  int elhm2 = 0;

  {
    const int m = j + ty;
    if (m < mcs_nrow) {
      // lig loop, ~30

      for (int i = tx; i < mcs_i2max[m]; i += bdx_mcs) { // x loop
        const int l = mcs[m].i2[i];
        const float dx = lig_x2[l] - mcs[m].x[i]; // do not use __LDG
        const float dy = lig_y2[l] - mcs[m].y[i];
        const float dz = lig_z2[l] - mcs[m].z[i];
        elhm1 += dx * dx + dy * dy + dz * dz;
        elhm2++;
      } // lig loop
    } // if (m < mcs_nrow)
  }

  BlockReduceSum_2D_2_d_2 (bdy_mcs, bdx_mcs, elhm1, elhm2);

  if (threadIdx.x < bdy_mcs) {
    const int m = j + threadIdx.x;
    if (m < mcs_nrow && elhm2 != 0)
      elhm += mcs_tcc[m] * sqrtf (elhm1 / (float) elhm2);
  }


 } // lhm loop

WarpReduceSum_1_d_2 (elhm);
if (threadIdx.x == 0)
  e_s[7] = logf (elhm / mcs_nrow);


