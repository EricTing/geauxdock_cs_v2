#include "hip/hip_runtime.h"
/*
#include <cstdlib>
#include <cstdio>

#include "dock.h"
#include "gpu.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
*/



// warp_id == bidx / warpSize == bidx >> WARPshift
// warp_land == bidx % warpSize == bidx & WARPmask

#ifndef WARPshift
#define WARPshift 5
#endif
#ifndef WARPmask
#define WARPmask 0b11111
#endif






__inline__ __device__
static void
BlockReduceSum_5_d_2 (float &a0, float &a1, float &a2, float &a3, float &a4)
{

  const int bidx = threadIdx.x;

  __shared__ float a0s[WARPperB];
  __shared__ float a1s[WARPperB];
  __shared__ float a2s[WARPperB];
  __shared__ float a3s[WARPperB];
  __shared__ float a4s[WARPperB];

  // 1st level warp reduction
#pragma unroll
  for (int stride = 16; stride > 0; stride >>= 1) {
    a0 += __shfl_xor (a0, stride);
    a1 += __shfl_xor (a1, stride);
    a2 += __shfl_xor (a2, stride);
    a3 += __shfl_xor (a3, stride);
    a4 += __shfl_xor (a4, stride);
  }

  {
    const int warp_lane = bidx & WARPmask;
    const int warp_id = bidx >> WARPshift;
    if (warp_lane == 0) {
      a0s[warp_id] = a0;
      a1s[warp_id] = a1;
      a2s[warp_id] = a2;
      a3s[warp_id] = a3;
      a4s[warp_id] = a4;
    }
  }

  __syncthreads ();
  a0 = (bidx < WARPperB) ? a0s[bidx] : 0.0f;
  a1 = (bidx < WARPperB) ? a1s[bidx] : 0.0f;
  a2 = (bidx < WARPperB) ? a2s[bidx] : 0.0f;
  a3 = (bidx < WARPperB) ? a3s[bidx] : 0.0f;
  a4 = (bidx < WARPperB) ? a4s[bidx] : 0.0f;

  // 2nd level warp reduction
#pragma unroll
  for (int stride = 16; stride > 0; stride >>= 1) {
    a0 += __shfl_xor (a0, stride);
    a1 += __shfl_xor (a1, stride);
    a2 += __shfl_xor (a2, stride);
    a3 += __shfl_xor (a3, stride);
    a4 += __shfl_xor (a4, stride);
  }
}


__inline__ __device__
static void
BlockReduceSum_1_d_2 (float &a0)
{

  const int bidx = threadIdx.x;

  __shared__ float a0s[WARPperB];

  // 1st level warp reduction
#pragma unroll
  for (int stride = 16; stride > 0; stride >>= 1)
    a0 += __shfl_xor (a0, stride);

  {
    const int warp_lane = bidx & WARPmask;
    const int warp_id = bidx >> WARPshift;
    if (warp_lane == 0)
      a0s[warp_id] = a0;
  }

  __syncthreads ();
  a0 = (bidx < WARPperB) ? a0s[bidx] : 0.0f;

  // 2nd level warp reduction
#pragma unroll
  for (int stride = 16; stride > 0; stride >>= 1)
    a0 += __shfl_xor (a0, stride);
}









__inline__ __device__
static void
WarpReduceSum_1_d_2 (float &a0)
{
  #pragma unroll
  for (int stride = 16; stride > 0; stride >>= 1) {
    a0 += __shfl_xor (a0, stride);
  }
}









__inline__ __device__
static void
BlockReduceSum_2D_2_d_2 (const int bdy, const int bdx, float &a0, int &a1)
{
  const int bidx = threadIdx.x;

  __shared__ float a0s[WARPperB];
  __shared__ int a1s[WARPperB];
  const int warp_lane = bidx & WARPmask;


  // warp reduction
#pragma unroll
  for (int stride = 16; stride > 0; stride >>= 1) {
    a0 += __shfl_xor (a0, stride);
    a1 += __shfl_xor (a1, stride);
  }

  {
    const int warp_id = bidx >> WARPshift;
    if (warp_lane == 0) {
      a0s[warp_id] = a0;
      a1s[warp_id] = a1;
    }
  }

  __syncthreads ();

  const int warp_x_per_b = bdx >> WARPshift;

  if (bidx < bdy) {
    a0 = 0.0f;
    a1 = 0;
#pragma unroll
    for (int s = 0; s < warp_x_per_b; ++s) {
      a0 += a0s[warp_x_per_b * bidx + s];
      a1 += a1s[warp_x_per_b * bidx + s];
    }
  }


}








__inline__ __device__
static void
BlockReduceSum_2D_1_d_2 (const int bdy, const int bdx, float &a0)
{
  const int bidx = threadIdx.x;

  __shared__ float a0s[WARPperB];
  const int warp_lane = bidx & WARPmask;


  // warp reduction
#pragma unroll
  for (int stride = 16; stride > 0; stride >>= 1) {
    a0 += __shfl_xor (a0, stride);
  }

  {
    const int warp_id = bidx >> WARPshift;
    if (warp_lane == 0) {
      a0s[warp_id] = a0;
    }
  }

  __syncthreads ();

  const int warp_x_per_b = bdx >> WARPshift;

  if (bidx < bdy) {
    a0 = 0;
#pragma unroll
    for (int s = 0; s < warp_x_per_b; ++s) {
      a0 += a0s[warp_x_per_b * bidx + s];
    }
  }


}
