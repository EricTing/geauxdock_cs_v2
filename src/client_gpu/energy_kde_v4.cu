#include "hip/hip_runtime.h"

float ekde = 0.0f;
ty = threadIdx.x / bdx_kde;
tx = threadIdx.x % bdx_kde;



for (int j = 0; j < lig_natom; j += bdy_kde) { // y loop, ~30
    float ekde1 = 0.0f;

    const int l = j + ty;
    //const int begin = kde_begin_idx[l];
    //const int end = kde_end_idx[l];

    if (l < lig_natom) {

//#pragma unroll 2
        for (int k = kde_begin_idx[l] + tx; k < kde_end_idx[l]; k += bdx_kde) { // x loop, ~400
            const float dx = lig_x3[l] - CUDA_LDG_D (kde->x[k]);
            const float dy = lig_y3[l] - CUDA_LDG_D (kde->y[k]);
            const float dz = lig_z3[l] - CUDA_LDG_D (kde->z[k]);
            const float kde_dst_pow2 = dx * dx + dy * dy + dz * dz;
            ekde1 += expf (enepara_kde2 * kde_dst_pow2);
        }
    }

    BlockReduceSum_2D_1_d_2 (bdy_kde, bdx_kde, ekde1);
    if (threadIdx.x < bdy_kde) {
        const int l = j + threadIdx.x;
        const float kde_sz = (float) (kde_end_idx[l] - kde_begin_idx[l]);
        if (l < lig_natom)
            ekde += ekde1 / kde_sz;
    }

} // lig loop

WarpReduceSum_1_d_2 (ekde);
if (threadIdx.x == 0)
    e_s[6] = ekde * enepara_kde3_inv;

