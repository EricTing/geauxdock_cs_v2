#include "hip/hip_runtime.h"
// add _LDG, slower


float elhm = 0.0f;
ty = threadIdx.x / bdx_mcs;
tx = threadIdx.x % bdx_mcs;


//Mcs *mcs_ = &complex->mcs[0];

// lhm loop, ~11
// unrolling outer loop does not help
//#pragma unroll 4
for (int j = 0; j < mcs_nrow; j += bdy_mcs) { // y loop
  float elhm1 = 0.0f;
  int elhm2 = 0;

  {
    const int m = j + ty;
    if (m < mcs_nrow) {
      // lig loop, ~30

      for (int l = tx; l < lig_natom; l += bdx_mcs) { // x loop
        if (CUDA_LDG_D (mcs[m].x[l]) != MCS_INVALID_COORD OROR1) {
          const float dx = lig_x2[l] - CUDA_LDG_D (mcs[m].x[l]);
          const float dy = lig_y2[l] - CUDA_LDG_D (mcs[m].y[l]);
          const float dz = lig_z2[l] - CUDA_LDG_D (mcs[m].z[l]);
          elhm1 += dx * dx + dy * dy + dz * dz;
          elhm2++;
        }

      } // lig loop
    } // if (m < mcs_nrow)
  }

  BlockReduceSum_2D_2_d_2 (bdy_mcs, bdx_mcs, elhm1, elhm2);

  if (threadIdx.x < bdy_mcs) {
    const int m = j + threadIdx.x;
    if (m < mcs_nrow && elhm2 != 0)
      elhm += mcs_tcc[m] * sqrtf (elhm1 / (float) elhm2);
  }


 } // lhm loop

WarpReduceSum_1_d_2 (elhm);
if (threadIdx.x == 0)
  e_s[7] = logf (elhm / mcs_nrow);

