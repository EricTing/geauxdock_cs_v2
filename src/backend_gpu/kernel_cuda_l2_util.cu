#include "hip/hip_runtime.h"
/*
#include <cstdlib>
#include <cstdio>

#include "geauxdock.h"
#include "gpu.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
 */




__forceinline__ __device__
float static
MyRand_d (hiprandState * curandstate_d)
{
    const int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState myseed = curandstate_d[gidx];
    float randdd = hiprand_uniform (&myseed);
    curandstate_d[gidx] = myseed;

    return randdd;
}




/*
minimizeing:
ceil ((float) data_x / x) * ceil ((float) data_y / y)
ST:
x = 32, 64, 128, 256, ... blockDim.x
y = blockDim.x / x

 */

# if 0
__device__
void
CalcThreadBlockShape (const int data_x, const int data_y, int & bdx, int & bdy)
{
    int var[11]; // at most 10 candidates, log2 (1024) = 10
    int imax = (int) log2 ((float) blockDim.x);

    for (int i = WARPshift; i <= imax; ++i) { // 32, 64, 128, 256, ... blockDim.x
        const int x = 1 << i;
        const int y = blockDim.x / x;
        var[i] = ceil ((float) data_x / x) * ceil ((float) data_y / y);
    }

    int iselect = WARPshift;
    for (int i = WARPshift; i <= imax; ++i) {
        if (var[iselect] >= var[i]) { // find the smallest (tend to adapt a larger i)
            iselect = i;
            //printf ("%03dx%03d: %3d     ", 1 << i, blockDim.x / (1 << i), var[i]);
        }
    }
    //printf ("\n");

    bdx = 1 << iselect;
    bdy = blockDim.x / bdx;

    //if (blockIdx.x == 0 && threadIdx.x == 0)
    //printf ("data: %3d x %3d     TB: %03dx%03d\n", data_x, data_y, bdx, bdy);
}
#endif




# if 1
__device__
void
CalcThreadBlockShape (const int data_x, const int data_y, int & bdx, int & bdy)
{
    int x_select = 32;
    int var_min = data_x * data_y / 32; // some rediculas large number

    for (int x = 32; x <= blockDim.x / 2; x += 32) { // 32, 64, 96, 128
        const int y = blockDim.x / x;
        if (blockDim.x % x == 0) {
            int var = ceil ((float) data_x / x) * ceil ((float) data_y / y);
            if (var < var_min) {
                x_select = x;
                var_min = var;
            }
            if (blockIdx.x == 0 && threadIdx.x == 0) {
                printf ("%03dx%03d: %3d\n", x, y, var);
            }
        }
    }

    bdx = x_select;
    bdy = blockDim.x / bdx;

    //if (blockIdx.x == 0 && threadIdx.x == 0)
    //printf ("data: %3d x %3d     TB: %03dx%03d\n", data_x, data_y, bdx, bdy);
}
#endif


