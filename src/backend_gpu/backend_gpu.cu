
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <ctime>

#include <geauxdock.h>
#include <size.h>
#include <toggle.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <geauxdock.h>
#include <toggle.h>
#include <util_print.h>

#include <yeah/cuda/runtime/wrapper.h>
#include <yeah/cuda/runtime/event.h>
#include <yeah/cuda/runtime/info.h>
#include <yeah/cuda/kernel/util.h>
#include <yeah/c/timing.h>
#include <yeah/cpp/timer.hpp>



#define CUDAASSERT(condition, code) \
    if (!(condition)) printf("Assertion failure. Code %s%n\n", code)

#include "kernel_cuda_l2_reduce_notemplate.cu"
#include "kernel_cuda_l2_util.cu"
#include "kernel_cuda_l1_initcurand.cu"
#include "kernel_cuda_l1_montecarlo.cu"



// GPU timer can't measure memory copy
// CPU timer can't measure mc kernel



void
InitCurand (hiprandState *s)
{
    srand (time (0));
    InitCurand_d <<< GD, BD >>> (s, rand ());
}



void
Dock (Complex *ch,
    Record *rh,
    Complex *cd,
    Record *rd,
    hiprandState *curandstate_d)
{
    const int steps_total = ch->mcpara.steps_total;
    const int steps_per_dump = ch->mcpara.steps_per_dump;
    const size_t record_sz = sizeof (Record) * ch->size.n_rep;


    yeah::Timer e[11];
    //yeah::cuda::EventSD e[11];

    CUDA_ERR (hipMemcpyAsync (cd, ch, sizeof (Complex), hipMemcpyHostToDevice));

    //GetPrintCudaFuncArributes ((void (*)) MonteCarlo_d, "MonteCarlo_d");
    //GetPrintCudaFuncArributes2 ((void (*)) MonteCarlo_d, "MonteCarlo_d", GD, BD, 0);


    e[10].Start ();


    e[3].Start ();
    MonteCarlo_d <<< GD, BD >>> (cd, rd, 0, 1, curandstate_d);
    CUDA_LAST_ERR ();
    e[3].Stop ();



    for (int s1 = 0; s1 < steps_total; s1 += steps_per_dump) {
        printf ("\t%d / %d \n", s1, steps_total);

        e[4].Start ();
        MonteCarlo_d <<< GD, BD >>> (cd, rd, s1, steps_per_dump, curandstate_d);
        CUDA_LAST_ERR ();
        //hipDeviceSynchronize();

        //yeah::Timer eeee;
        //eeee.Start ();
        CUDA_ERR (hipMemcpy (rh, rd, record_sz, hipMemcpyDeviceToHost));
        // eeee.Stop ();
        // printf ("launcher: time of memory copy D2H %f\n", eeee.Span());
        e[4].Stop ();
//#include <kernel_dump.C>
    }

    Record *record = rh;
#include <kernel_print.C>


    e[10].Stop ();


#include <kernel_print_timer.C>
    //PrintSummary (ch);
#include <kernel_print_benchmark.C>

}



